#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <memory.h>

#define MAX_BLOCKS 0x100
#define MAX_FSIZE 0x10000000
#define BYTE_LIM 0x100

#define MIN(x,y) (x) < (y) ? (x) : (y)

static int parse_args(int argc, char** argv);
static int read_file(const char*, uint8_t**, int);
void print_stats(const uint32_t* stats, int num_threads, int num_blocks);

static uint32_t flp2(uint32_t x) {
    x = x | (x >> 1);
    x = x | (x >> 2);
    x = x | (x >> 4);
    x = x | (x >> 8);
    x = x | (x >> 16);
    return x - (x >> 1);
}

static hipError_t bruteforce(uint8_t* buf, uint32_t* stats, int num_threads, int num_chunks);

__global__ void bruteforce_kernel(uint8_t* buf, uint32_t* g_stats, int num_chunks) {
    uint32_t local_stats[BYTE_LIM] = { 0 };
    constexpr int32_t lower_lim = 0x20; // ascii space
    constexpr int32_t upper_lim = 0x7e; // ascii ~
    int curr_chunk = blockIdx.x;
    while (curr_chunk < num_chunks) {
        const int32_t ch = (int32_t)buf[threadIdx.x + curr_chunk * blockDim.x];
        for (int32_t j = 0; j < BYTE_LIM; j++) {
            const int32_t decrypted = ch ^ j;
            const int32_t greater = decrypted - lower_lim;
            const int32_t less = upper_lim - decrypted;
            const uint32_t inside = (~((uint32_t)greater | (uint32_t)less)) >> 31;
            local_stats[j] += inside;
        }
        curr_chunk += gridDim.x;
    }
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int offset = tid * BYTE_LIM;
    for (int i = 0; i < BYTE_LIM; i++)
        g_stats[i + offset] = local_stats[i];
}

__global__ void reduction_kernel(uint32_t* g_stats) {
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int offset = tid * BYTE_LIM;
    const int tid_2 = (gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
    const int offset_2 = tid_2 * BYTE_LIM;
    for (int i = 0; i < BYTE_LIM; i++)
        g_stats[i + offset] += g_stats[i + offset_2];
}

int main(int argc, char** argv) {
    const int num_threads = parse_args(argc, argv);
    if (num_threads <= 0)
        return -1;

    uint8_t* buf = nullptr;

    const int num_chunks = read_file(argv[1], &buf, num_threads);
    if (num_chunks <= 0) {
        return -1;
    }

    int tolerance = 0;

    if (argc < 4) {
        puts("No tolerance value supplied. Zero tolerance will be used.");
    } else {
        tolerance = atoi(argv[3]);
        printf("Tolerance = %d chars.\n", tolerance);
    }
    if (num_chunks <= tolerance) {
        printf("Max tolerance value for this file is: %d\n", num_chunks - 2);
        return -1;
    }

    // stats[key_id][char+id]
    uint32_t* stats = (uint32_t*)malloc((size_t)(num_threads * BYTE_LIM * sizeof(uint32_t)));
    if (stats == nullptr) {
        puts("Error allocationg stats buffer!");
        free(buf);
        return -1;
    }

    hipError_t cudaStatus = bruteforce(buf, stats, num_threads, num_chunks);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        free(buf);
        free(stats);
        return 1;
    }

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        free(buf);
        free(stats);
        return 1;
    }

    print_stats(stats, num_threads, num_chunks - tolerance);

    free(buf);
    free(stats);
    return 0;
}

static hipError_t bruteforce(uint8_t* buf, uint32_t* stats, int num_threads, int num_chunks)
{
    uint8_t* dev_buf = nullptr;
    uint32_t* dev_stats = nullptr;
    hipError_t cudaStatus;

    hipDeviceProp_t prop;
    int dev;
    memset(&prop, 0, sizeof(hipDeviceProp_t));
    prop.major = 1;
    prop.minor = 0;
    cudaStatus = hipChooseDevice(&dev, &prop);
    // Choose which GPU to run on, change this on a multi-GPU system.
    //cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }
    hipGetDeviceProperties(&prop, dev);
    const int max_blocks = MIN(flp2(prop.multiProcessorCount * 2 * prop.maxBlocksPerMultiProcessor), MAX_BLOCKS);   // MAX_BLOCKS is experimental value

    int num_blocks = MIN(num_chunks, max_blocks);
    const size_t size = num_threads * num_chunks;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);


    cudaStatus = hipMalloc((void**)&dev_buf, size * sizeof(uint8_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_stats, (size_t)(num_blocks * num_threads) * BYTE_LIM * sizeof(uint32_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_buf, buf, size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    bruteforce_kernel << <num_blocks, num_threads >> > (dev_buf, dev_stats, num_chunks);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "bruteforce_kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // reduction stage
    num_blocks /= 2;
    while (num_blocks) {
        reduction_kernel << <num_blocks, num_threads >> > (dev_stats);
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "reduction_kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            goto Error;
        }
        num_blocks /= 2;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Device synchronization failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(stats, dev_stats, (size_t)(num_threads * BYTE_LIM * sizeof(uint32_t)), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Time to calculate: %3.1f ms\n", elapsedTime);

Error:
    hipFree(dev_buf);
    hipFree(dev_stats);

    return cudaStatus;
}

static int parse_args(int argc, char** argv) {
    if (argc < 3) {
        puts("Provide the filename and key size (either 32, 64, 128 or 256 bytes)!");
        return -1;
    }
    int num_t = atoi(argv[2]);
    if ((num_t != 32) && (num_t != 64) && (num_t != 128) && (num_t != 256)) {
        puts("Key size must be either 32, 64, 128 or 256 bytes!");
        num_t = -1;
    }
    return num_t;
}

static int read_file(const char* fname, uint8_t** buf, int num_threads)
{
    FILE* file = fopen(fname, "rb");
    if (!file) {
        puts("Error opening file!");
        return -1;
    }
    fseek(file, 0L, SEEK_END);
    const size_t file_size = ftell(file);
    if (file_size < num_threads) {
        puts("File size to small for any meaningfull processing!");
        fclose(file);
        return -1;
    }
    rewind(file);
    if (file_size > MAX_FSIZE) {
        puts("File exceeding maximum size!");
        fclose(file);
        return -1;
    }
    const int num_chunks = (int)(flp2((uint32_t)(file_size / num_threads)));  // num_blocks should be power of two for the reduction step
    const int read_size = num_chunks * num_threads;
    if (!(*buf = (uint8_t*)malloc(read_size))) {
        puts("Buffer allocation failed!");
        fclose(file);
        return -1;
    }
    if (fread((void*)*(buf), 1, read_size, file) != read_size) {
        puts("Error reading file!");
        free(*buf);
        fclose(file);
        return -1;
    }
    fclose(file);

    return num_chunks;
}

void print_stats(const uint32_t* stats, int num_threads, int num_blocks)
{
    printf("%d-byte xor encryption key stats:\n", num_threads);
    for (uint16_t i = 0u; i < num_threads; i++) {
        printf("For byte #%d possible char codes are:\t", i);
        for (uint16_t j = 0u; j < BYTE_LIM; j++) {
            if (stats[i * BYTE_LIM + j] >= num_blocks) {
                printf("%x ", j);
            }
            //printf("%d ", stats[i * BYTE_LIM + j]);
        }
        puts("");
    }
}
